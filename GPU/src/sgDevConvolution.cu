#include "hip/hip_runtime.h"
#include "Convolution.cuh"
// Define the device pointer to the callback routine. The host code will fetch this and pass it to CUFFT
#ifndef WIN64
__device__ hipfftCallbackLoadC myOwnCallbackPtr = cbComplexPointwiseMul;
#endif
void findBlockSize(long long iFrames, int M, size_t *blockSize, int *blockNum) {
	/*Finding block size/number*/

	int myExp = ceil(log2((float)(iFrames + M)));
	while (pow(2, myExp) > INT_MAX) {
		myExp--;
	}
	size_t smallerBlockSize = pow(2, myExp);
	*blockNum = 1;
	size_t workspace;
	CHECK_CUFFT_ERRORS(hipfftEstimate1d(smallerBlockSize, HIPFFT_R2C, 2, &workspace));

	/*Look for block size worth with 2 complex arrays
	Multiply by 4 to leave some room*/
	while (getFreeSize() < workspace + (smallerBlockSize / 2 + 1) * 8L * 4L) {
		myExp--;
		smallerBlockSize = pow(2, myExp);
		(*blockNum)++;
		CHECK_CUFFT_ERRORS(hipfftEstimate1d(smallerBlockSize, HIPFFT_R2C, 2, &workspace));
	}

	fprintf(stderr, "blockSize: %i\t numBlocks: %i\n", smallerBlockSize, *blockNum);
	*blockSize = smallerBlockSize;
}

void mismatchedConvolve(passable *p) {
	flags flag = p->type;
	long long paddedSize = p->paddedSize;
	float *d_ibuf = p->input->d_buf;
	float *d_rbuf = p->reverb->d_buf;

	/*Create forward FFT plan*/
	hipfftHandle plan;
	CHECK_CUFFT_ERRORS(hipfftCreate(&plan));
	CHECK_CUFFT_ERRORS(hipfftPlan1d(&plan, paddedSize, HIPFFT_R2C, 1));

	/*Create inverse FFT plan*/
	hipfftHandle outplan;
	CHECK_CUFFT_ERRORS(hipfftCreate(&outplan));
	CHECK_CUFFT_ERRORS(hipfftPlan1d(&outplan, paddedSize, HIPFFT_C2R, 1));

	/*Transform Input Signal*/
	CHECK_CUFFT_ERRORS(hipfftExecR2C(plan, (hipfftReal *)d_ibuf, (hipfftComplex*)d_ibuf));
	if (flag == stereo_mono) {
		Print("Transforming Ch 2 of input\n");
		CHECK_CUFFT_ERRORS(hipfftExecR2C(plan, (hipfftReal *)d_ibuf + paddedSize, (hipfftComplex*)d_ibuf + paddedSize / 2 + 1));
	}
	/*Transform Filter Signal*/
	CHECK_CUFFT_ERRORS(hipfftExecR2C(plan, (hipfftReal*)d_rbuf, (hipfftComplex*)d_rbuf));
	if (flag == mono_stereo) {
		Print("Transforming Ch 2 of reverb\n");
		CHECK_CUFFT_ERRORS(hipfftExecR2C(plan, (hipfftReal *)d_rbuf + paddedSize, (hipfftComplex*)d_rbuf + paddedSize / 2 + 1));
	}
#if defined WIN64 || CB == 0
	/*NO CB VERSION*/

	/*CONVOLUTION*/
	int blockSize = 256;
	int numBlocks = (paddedSize / 2 + 1 + blockSize - 1) / blockSize;
	if (flag == mono_stereo) {
		Print("Convolving & Inverse Transforming for stereo reverb\n");
		ComplexPointwiseMul << < numBlocks, blockSize >> > ((hipfftComplex*)d_rbuf, (hipfftComplex*)d_ibuf, paddedSize / 2 + 1);
		ComplexPointwiseMul << < numBlocks, blockSize >> > ((hipfftComplex*)d_rbuf + paddedSize / 2 + 1, (hipfftComplex*)d_ibuf, paddedSize / 2 + 1);
	}
	else {
		ComplexPointwiseMul << < numBlocks, blockSize >> > ((hipfftComplex*)d_ibuf, (hipfftComplex*)d_rbuf, paddedSize / 2 + 1);
		ComplexPointwiseMul << < numBlocks, blockSize >> > ((hipfftComplex*)d_ibuf + paddedSize / 2 + 1, (hipfftComplex*)d_rbuf, paddedSize / 2 + 1);
	}
#else
	/*Copy over the host copy of callback function*/
	hipfftCallbackLoadC hostCopyOfCallbackPtr;
	checkCudaErrors(hipMemcpyFromSymbol(&hostCopyOfCallbackPtr, HIP_SYMBOL(myOwnCallbackPtr), sizeof(hostCopyOfCallbackPtr)));

	/*Associate the load callback with the plan*/
	if (flag == stereo_mono) {
		CHECK_CUFFT_ERRORS(hipfftXtSetCallback(outplan, (void **)&hostCopyOfCallbackPtr, HIPFFT_CB_LD_COMPLEX,
			(void **)&d_rbuf));
	}
	else {
		CHECK_CUFFT_ERRORS(hipfftXtSetCallback(outplan, (void **)&hostCopyOfCallbackPtr, HIPFFT_CB_LD_COMPLEX,
			(void **)&d_ibuf));
	}
#endif
	if (flag == stereo_mono) {
		CHECK_CUFFT_ERRORS(hipfftExecC2R(outplan, (hipfftComplex*)d_ibuf, d_ibuf));
		CHECK_CUFFT_ERRORS(hipfftExecC2R(outplan, (hipfftComplex*)d_ibuf + paddedSize / 2 + 1, d_ibuf + paddedSize));
	}
	else {
		CHECK_CUFFT_ERRORS(hipfftExecC2R(outplan, (hipfftComplex*)d_rbuf, d_rbuf));
		CHECK_CUFFT_ERRORS(hipfftExecC2R(outplan, (hipfftComplex*)d_rbuf + paddedSize / 2 + 1, d_rbuf + paddedSize));
	}
	checkCudaErrors(hipfftDestroy(plan));
	checkCudaErrors(hipfftDestroy(outplan));
}

void convolve(float *d_ibuf, float *d_rbuf, long long paddedSize) {
	/*Create forward FFT plan*/
	hipfftHandle plan;
	CHECK_CUFFT_ERRORS(hipfftCreate(&plan));
	CHECK_CUFFT_ERRORS(hipfftPlan1d(&plan, paddedSize, HIPFFT_R2C, 1));

	/*Create inverse FFT plan*/
	hipfftHandle outplan;
	CHECK_CUFFT_ERRORS(hipfftCreate(&outplan));
	CHECK_CUFFT_ERRORS(hipfftPlan1d(&outplan, paddedSize, HIPFFT_C2R, 1));

	/*Transform Complex Signal*/
	CHECK_CUFFT_ERRORS(hipfftExecR2C(plan, (hipfftReal*)d_ibuf, (hipfftComplex*)d_ibuf));

	/*Transform Filter Signal*/
	CHECK_CUFFT_ERRORS(hipfftExecR2C(plan, (hipfftReal*)d_rbuf, (hipfftComplex*)d_rbuf));

#if defined WIN64 || CB == 0
	/*NO CB VERSION*/
	/*CONVOLUTION*/
	int blockSize = 256;
	int numBlocks = (paddedSize + blockSize - 1) / blockSize;

	ComplexPointwiseMul << < numBlocks, blockSize >> > ((hipfftComplex*)d_ibuf, (hipfftComplex*)d_rbuf, paddedSize / 2 + 1);
	getLastCudaError("Kernel execution failed [ ComplexPointwiseMul]");
#else
	/*Copy over the host copy of callback function*/
	hipfftCallbackLoadC hostCopyOfCallbackPtr;
	checkCudaErrors(hipMemcpyFromSymbol(&hostCopyOfCallbackPtr, HIP_SYMBOL(myOwnCallbackPtr),
		sizeof(hostCopyOfCallbackPtr)));

	/*Associate the load callback with the plan*/
	CHECK_CUFFT_ERRORS(hipfftXtSetCallback(outplan, (void **)&hostCopyOfCallbackPtr, HIPFFT_CB_LD_COMPLEX,
		(void **)&d_rbuf));

#endif
	CHECK_CUFFT_ERRORS(hipfftExecC2R(outplan, (hipfftComplex*)d_ibuf, (hipfftReal*)d_ibuf));
	checkCudaErrors(hipfftDestroy(plan));
	checkCudaErrors(hipfftDestroy(outplan));
}
/*Assumes that d_buf contains paddedSize * 2 elements.
Input is in first half, filter is in second half, and both are padded*/
void convolveBatched(float *d_buf, long long paddedSize) {
	float *d_rbuf = d_buf + paddedSize + 2;
	/*Create forward FFT plan*/
	hipfftHandle plan;
	CHECK_CUFFT_ERRORS(hipfftCreate(&plan));
	/*hipfftResult hipfftPlanMany(hipfftHandle *plan, int rank, int *n,
		int *inembed, int istride, int idist,
		int *onembed, int ostride, int odist,
		hipfftType type, int batch);*/
		/*stride = skip length. Ex 1 = every element, 2 = every other element*/
			/*use for interleaving???*/
		/*idist/odist is space between batches of transforms*/
			/*need to check if odist is in terms of complex numbers or floats*/
		/*inembed/onembed are for 2D/3D, num elements per dimension*/
	int n = paddedSize;
	CHECK_CUFFT_ERRORS(
		hipfftPlanMany(&plan, 1, &n,
			&n, 1, n + 2,
			&n, 1, n / 2 + 1,
			HIPFFT_R2C, 2)
	)

		/*Create inverse FFT plan*/
		hipfftHandle outplan;
	CHECK_CUFFT_ERRORS(hipfftCreate(&outplan));
	CHECK_CUFFT_ERRORS(hipfftPlan1d(&outplan, paddedSize, HIPFFT_C2R, 1));

	/*Transform Complex Signal*/
	CHECK_CUFFT_ERRORS(hipfftExecR2C(plan, (hipfftReal*)d_buf, (hipfftComplex*)d_buf));

#if defined WIN64 || CB == 0
	/*NO CB VERSION*/
	/*CONVOLUTION*/
	int blockSize = 256;
	int numBlocks = (paddedSize + blockSize - 1) / blockSize;

	ComplexPointwiseMul << < numBlocks, blockSize >> > ((hipfftComplex*)d_buf, (hipfftComplex*)d_rbuf, paddedSize / 2 + 1);
	getLastCudaError("Kernel execution failed [ ComplexPointwiseMul]");
#else
	/*Copy over the host copy of callback function*/
	hipfftCallbackLoadC hostCopyOfCallbackPtr;
	checkCudaErrors(hipMemcpyFromSymbol(&hostCopyOfCallbackPtr, HIP_SYMBOL(myOwnCallbackPtr),
		sizeof(hostCopyOfCallbackPtr)));

	/*Associate the load callback with the plan*/
	CHECK_CUFFT_ERRORS(hipfftXtSetCallback(outplan, (void **)&hostCopyOfCallbackPtr, HIPFFT_CB_LD_COMPLEX,
		(void **)&d_rbuf));

#endif
	CHECK_CUFFT_ERRORS(hipfftExecC2R(outplan, (hipfftComplex*)d_buf, (hipfftReal*)d_buf));
	checkCudaErrors(hipfftDestroy(plan));
	checkCudaErrors(hipfftDestroy(outplan));
}

void overlapAdd(float *d_ibuf, hipfftComplex *d_rbuf, long long iFrames, long long M,
	long long blockSize, int blockNum, hipfftHandle plan, hipfftHandle outplan) {
	float *d_block;
	long long L = blockSize - M;

	int numThreads = 256;
	int numBlocks = (M + numThreads - 1) / numThreads;

	checkCudaErrors(hipMalloc(&d_block, (blockSize / 2 + 1) * sizeof(hipfftComplex)));

	for (int blockNo = 0; blockNo < blockNum; blockNo++) {
		long long cpyAmount = L;
		if (blockNo == blockNum && iFrames != cpyAmount) {
			cpyAmount = iFrames % L;
		}
		/*1/5/11/17 - Copy buf(N * L, L) -> sig[0]. cpyAmount becomes R at the end. N = 0 initially*/
		//fprintf(stderr, "Copy(block, obuf[%'i], %'i)\n", L * blockNo, cpyAmount);
		checkCudaErrors(hipMemcpy(d_block, &d_ibuf[L * blockNo], cpyAmount * sizeof(float), hipMemcpyDeviceToDevice));
		if (blockNo != 0) {
			/*6/12/18 - Copy sig(L, M) -> buf[N * L]*/
			//fprintf(stderr, "Copy(obuf[%'i], block[%'i], %'i)\n", L * blockNo, L, M);
			checkCudaErrors(hipMemcpy(&d_ibuf[L * blockNo], &d_block[L], M * sizeof(float), hipMemcpyDeviceToDevice));
		}

		/*2/7/13/19 - Pad sig(L, M) with 0's, cpyAmount becomes R at the end*/
		fillWithZeroes(&d_block, cpyAmount, blockSize);

		/*Transform signal*/
		CHECK_CUFFT_ERRORS(hipfftExecR2C(plan, (hipfftReal *)d_block, (hipfftComplex*)d_block));

#if defined WIN64 || CB == 0
		/*CONVOLUTION*/
		/*3/8/14/20*/
		numBlocks = (blockSize / 2 + numThreads) / numThreads;
		ComplexPointwiseMul << < numBlocks, numThreads >> > ((hipfftComplex*)d_block,
			(hipfftComplex*)d_rbuf, blockSize / 2 + 1);
		getLastCudaError("Kernel execution failed [ ComplexPointwiseMul]");
#endif
		/*IFFT*/
		CHECK_CUFFT_ERRORS(hipfftExecC2R(outplan, (hipfftComplex*)d_block, (hipfftReal*)d_block));
		if (blockNo != 0) {
			/* 9/15/21 - Point-wise add sig(0,M) + buf[N*L]*/
			PointwiseAdd << <numBlocks, numThreads >> > ((float*)d_block, &d_ibuf[blockNo * L], M);
		}
		checkCudaErrors(hipDeviceSynchronize());
		/*Corner case where only one block*/
		if (blockNo == 0 && blockNo == blockNum - 1) {
			checkCudaErrors(hipMemcpy(d_ibuf, d_block, (cpyAmount + M) * sizeof(float), hipMemcpyDeviceToDevice));
			break;
		}
		/*Initial case*/
		if (blockNo == 0) {
			/*4 - Copy sig(0,L) -> buf[0]*/
			checkCudaErrors(hipMemcpy(d_ibuf, d_block, L * sizeof(float), hipMemcpyDeviceToDevice));
		}
		/*Last case*/
		if (blockNo == blockNum - 1) {
			//fprintf(stderr, "Copy(obuf[%'i], block[%'i], %'i)\n", blockNo * L + M, M, cpyAmount);
			checkCudaErrors(hipMemcpy(&d_ibuf[blockNo * L + M], &d_block[M], cpyAmount * sizeof(float), hipMemcpyDeviceToDevice));
		}
		/*Every other case*/
		if (blockNo != 0 && blockNo < blockNum) {
			/*10/16 - Copy sig(M, L-M) -> buf[N * L + M]*/
			checkCudaErrors(hipMemcpy(&d_ibuf[blockNo * L + M], &d_block[M], (L - M) * sizeof(float), hipMemcpyDeviceToDevice));
		}
	}
	checkCudaErrors(hipFree(d_block));
}

float *blockConvolution(passable *p) {
	float *d_ibuf = p->input->d_buf;
	float *rbuf = p->reverb->buf;
	hipfftComplex *d_filter_complex;
	float *d_obuf = d_ibuf, *obuf;
	long long rFrames = p->reverb->frames;
	long long iFrames = p->input->frames;
	long long oFrames = rFrames + iFrames - 1;
	flags flag = p->type;
	int oCh = flag == mono_mono ? 1 : 2;
	float minmax, minmax2;
	hipEvent_t start, stop;


	int M = rFrames - 1;

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	size_t blockSize = iFrames;
	int blockNum = 0;

	/*Find block size and store in blockSize and blockNum*/
	findBlockSize(iFrames, M, &blockSize, &blockNum);

	/*Allocating memory for output*/
	Print("Allocating memory for output\n");
	checkCudaErrors(hipHostMalloc((void**)&obuf, oFrames * oCh * sizeof(float)));


	/*Find peak of input signal*/
	Print("Finding peak of input signal\n");
	minmax = DExtrema(d_ibuf, oFrames * p->input->channels);

	/*TRANSFORMING FILTER*/
	/*Allocating Memory*/
	Print("Allocating memory\n");
	int ch = p->reverb->channels;
	checkCudaErrors(hipMalloc(&d_filter_complex, (blockSize / 2 + 1) * ch * sizeof(hipfftComplex)));

	/*Block/Thread sizes for kernels*/
	int numThreads = 256;
	int numBlocks = (blockSize + 2 - rFrames + numThreads - 1) / numThreads;
	hipStream_t stream[4];
	for (int i = 0; i < 4; i++) {
		checkCudaErrors(hipStreamCreate(&stream[i]));
	}
	/* Copy over filter */
	Print("Copying over filter\n");
	FillWithZeros << <numBlocks, numThreads, 0, stream[0] >> > ((float*)d_filter_complex, rFrames, blockSize + 2);
	if (ch == 2) {
		FillWithZeros << <numBlocks, numThreads, 0, stream[1] >> > ((float*)d_filter_complex + blockSize + 2,
			rFrames, blockSize * 2 + 4);
		checkCudaErrors(hipMemcpyAsync((float*)d_filter_complex + blockSize + 2,
			rbuf + rFrames, rFrames * sizeof(float), hipMemcpyHostToDevice, stream[2]));
	}
	checkCudaErrors(hipMemcpyAsync((float*)d_filter_complex, rbuf,
		rFrames * sizeof(float), hipMemcpyHostToDevice, stream[3]));


	/*Create cuFFT plan*/
	Print("Creating FFT plans\n");
	hipfftHandle plan;
	CHECK_CUFFT_ERRORS(hipfftCreate(&plan));
	CHECK_CUFFT_ERRORS(hipfftPlan1d(&plan, blockSize, HIPFFT_R2C, 1));

	/*Plans*/
	hipfftHandle outplan;
	CHECK_CUFFT_ERRORS(hipfftCreate(&outplan));
	CHECK_CUFFT_ERRORS(hipfftPlan1d(&outplan, blockSize, HIPFFT_C2R, 1));


#if defined WIN64 || CB == 0
#else
	/*Create host pointer to CB Function*/
	hipfftCallbackLoadC hostCopyOfCallbackPtr;
	checkCudaErrors(hipMemcpyFromSymbol(&hostCopyOfCallbackPtr, HIP_SYMBOL(myOwnCallbackPtr), sizeof(hostCopyOfCallbackPtr)));

	/*Associate the load callback with the plan*/
	CHECK_CUFFT_ERRORS(hipfftXtSetCallback(outplan, (void **)&hostCopyOfCallbackPtr,
		HIPFFT_CB_LD_COMPLEX, (void **)&d_filter_complex));
#endif	

	for (int i = 0; i < 4; i++) {
		checkCudaErrors(hipStreamSynchronize(stream[i]));
	}

	checkCudaErrors(hipHostFree(rbuf));

	/*Transform Filter*/
	Print("Transforming filter\n");
	CHECK_CUFFT_ERRORS(hipfftExecR2C(plan, (hipfftReal *)d_filter_complex, (hipfftComplex*)d_filter_complex));
	if (ch == 2) {
		CHECK_CUFFT_ERRORS(hipfftExecR2C(plan, (hipfftReal *)d_filter_complex + blockSize + 2, (hipfftComplex*)d_filter_complex + blockSize / 2 + 1));
	}

	/*Convolving*/
	if (flag == mono_mono) {
		Print("mono_mono Convolving\n");
		overlapAdd(d_obuf, d_filter_complex, iFrames, M, blockSize, blockNum, plan, outplan);
	}
	else if (flag == stereo_stereo) {
		Print("stereo_stereo Convolving\n");
		overlapAdd(d_obuf, d_filter_complex, iFrames, M, blockSize, blockNum, plan, outplan);
		overlapAdd(d_obuf + oFrames, d_filter_complex + blockSize / 2 + 1,
			iFrames, M, blockSize, blockNum, plan, outplan);
	}
	else if (flag == stereo_mono) {
		Print("stereo_mono Convolving\n");
		overlapAdd(d_obuf, d_filter_complex, iFrames, M, blockSize, blockNum, plan, outplan);
		overlapAdd(d_obuf + oFrames, d_filter_complex, iFrames, M, blockSize, blockNum, plan, outplan);
	}
	else {
		Print("mono_stereo Convolving\n");
		checkCudaErrors(hipMemcpy(d_obuf + oFrames, d_obuf, oFrames * sizeof(float), hipMemcpyDeviceToDevice));
		overlapAdd(d_obuf, d_filter_complex, iFrames, M, blockSize, blockNum, plan, outplan);
		overlapAdd(d_obuf + oFrames, d_filter_complex + blockSize / 2 + 1,
			iFrames, M, blockSize, blockNum, plan, outplan);
	}
	checkCudaErrors(hipFree(d_filter_complex));
	CHECK_CUFFT_ERRORS(hipfftDestroy(plan));
	CHECK_CUFFT_ERRORS(hipfftDestroy(outplan));

	/*Find peak of output*/
	Print("Find peak of output\n");
	minmax2 = DExtrema(d_obuf, oFrames * oCh);

	float scale = minmax / minmax2;
	long long end = oFrames * oCh;
	fprintf(stderr, "end: %lli\n", end);
	/*Block/Thread sizes for kernels*/
	blockSize = 512;
	numBlocks = (end + blockSize - 1) / blockSize;
	// RealFloatScale << < numBlocks, blockSize>> > (d_obuf, end, scale);
	// checkCudaErrors(hipMemcpy(obuf, d_obuf, end * sizeof(float), hipMemcpyDeviceToHost));

	/*Asynchronous copy & scale */
	const int nStreams = 4;
	int streamSize = (end + nStreams - 1) / nStreams;
	int streamBytes = streamSize * sizeof(float);
	numBlocks = (streamSize + blockSize - 1) / blockSize;

	Print("Scaling and copying\n");
	for (int i = 0; i < nStreams; ++i) {
		long long offset = i * streamSize;
		/*Run scale kernel*/
		RealFloatScaleConcurrent << < numBlocks, blockSize, 0, stream[i] >> > (d_obuf, end, streamSize, scale, offset);
		/*Copy device memory to host asynchronously*/
		if (i == nStreams - 1) {
			streamBytes = sizeof(float) * (end - offset);
		}
		checkCudaErrors(hipMemcpyAsync(&obuf[offset], &d_obuf[offset], streamBytes, hipMemcpyDeviceToHost, stream[i]));
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	fprintf(stderr, "Time for GPU convolution: %f ms\n", milliseconds);

	checkCudaErrors(hipFree(d_obuf));
	return obuf;
}
float *convolution(passable *p) {
	float *d_ibuf = p->input->d_buf;
	float *d_rbuf = p->reverb->d_buf;
	float *d_obuf = d_ibuf;
	float *obuf;
	flags flag = p->type;
	int oCh = flag == mono_mono ? 1 : 2;
	long long paddedSize = p->paddedSize;
	float minmax, minmax2;
	hipEvent_t start, stop;
	//printMe(p);

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

	/*Allocating host memory for output*/
	Print("Allocating host memory for output\n");
	checkCudaErrors(hipHostMalloc((void**)&obuf, paddedSize * oCh * sizeof(float)));

	/*Find peak of input signal*/
	Print("Finding peak of input signal\n");
	minmax = DExtrema(d_ibuf, paddedSize * oCh);

	/*Convolving*/
	if (flag == mono_mono) {
		Print("mono_mono Convolving\n");
		convolve(d_ibuf, d_rbuf, paddedSize);
		//convolveBatched(d_ibuf, paddedSize); // not doing batched because it is very slightly slower (~20 ms)
	}
	else if (flag == stereo_stereo) {
		Print("stereo_stereo Convolving\n");
		convolve(d_ibuf, d_rbuf, paddedSize);
		convolve(d_ibuf + paddedSize, d_rbuf + paddedSize, paddedSize);
	}
	else {
		mismatchedConvolve(p);
		if (flag == mono_stereo) {
			d_obuf = d_rbuf;
		}
	}

	/*Find peak of output*/
	Print("Find peak of output\n");
	minmax2 = DExtrema(d_obuf, paddedSize * oCh);

	float scale = minmax / minmax2;
	long long end = paddedSize * oCh;

	/*Block/Thread sizes for kernels*/
	int blockSize = 512;
	int numBlocks = (end + blockSize - 1) / blockSize;

	/*Asynchronous copy & scale */
	const int nStreams = 4;
	int streamSize = (end + nStreams - 1) / nStreams;
	int streamBytes = streamSize * sizeof(float);
	numBlocks = (streamSize + blockSize - 1) / blockSize;

	/*Create streams*/
	Print("Creating streams\n");
	hipStream_t stream[nStreams];
	for (int i = 0; i < nStreams; ++i) {
		checkCudaErrors(hipStreamCreate(&stream[i]));
	}
	Print("Scaling and copying\n");
	for (int i = 0; i < nStreams; ++i) {
		long long offset = i * streamSize;
		/*Run scale kernel*/
		RealFloatScaleConcurrent << < numBlocks, blockSize, 0, stream[i] >> > (d_obuf, end, streamSize, scale, offset);
		/*Copy device memory to host asynchronously*/
		if (i == nStreams - 1) {
			streamBytes = sizeof(float) * (end - offset);
		}
		checkCudaErrors(hipMemcpyAsync(&obuf[offset], &d_obuf[offset], streamBytes, hipMemcpyDeviceToHost, stream[i]));
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	fprintf(stderr, "Time for GPU convolution: %f ms\n", milliseconds);

	checkCudaErrors(hipFree(d_ibuf));
	checkCudaErrors(hipFree(d_rbuf));
	return obuf;
}
void blockProcess(passable* p) {
	float* d_ibuf = p->input->d_buf;
	float* rbuf = p->reverb->buf;
	hipfftComplex* d_filter_complex;
	float* d_obuf = d_ibuf;
	long long rFrames = p->reverb->frames;
	long long iFrames = p->input->frames;
	long long oFrames = rFrames + iFrames - 1;
	flags flag = p->type;
	int M = rFrames - 1;
	size_t blockSize = iFrames;
	int blockNum = 0;

	/*Find block size and store in blockSize and blockNum*/
	findBlockSize(iFrames, M, &blockSize, &blockNum);

	/*TRANSFORMING FILTER*/
	/*Allocating Memory*/
	Print("Allocating memory\n");
	int ch = p->reverb->channels;
	checkCudaErrors(hipMalloc(&d_filter_complex, (blockSize / 2 + 1) * ch * sizeof(hipfftComplex)));

	/*Block/Thread sizes for kernels*/
	int numThreads = 256;
	int numBlocks = (blockSize + 2 - rFrames + numThreads - 1) / numThreads;
	hipStream_t stream[4];
	for (int i = 0; i < 4; i++) {
		checkCudaErrors(hipStreamCreate(&stream[i]));
	}
	/* Copy over filter */
	Print("Copying over filter\n");
	FillWithZeros << <numBlocks, numThreads, 0, stream[0] >> > ((float*)d_filter_complex, rFrames, blockSize + 2);
	if (ch == 2) {
		FillWithZeros << <numBlocks, numThreads, 0, stream[1] >> > ((float*)d_filter_complex + blockSize + 2,
			rFrames, blockSize * 2 + 4);
		checkCudaErrors(hipMemcpyAsync((float*)d_filter_complex + blockSize + 2,
			rbuf + rFrames, rFrames * sizeof(float), hipMemcpyHostToDevice, stream[2]));
	}
	checkCudaErrors(hipMemcpyAsync((float*)d_filter_complex, rbuf,
		rFrames * sizeof(float), hipMemcpyHostToDevice, stream[3]));


	/*Create cuFFT plan*/
	Print("Creating FFT plans\n");
	hipfftHandle plan;
	CHECK_CUFFT_ERRORS(hipfftCreate(&plan));
	CHECK_CUFFT_ERRORS(hipfftPlan1d(&plan, blockSize, HIPFFT_R2C, 1));

	/*Plans*/
	hipfftHandle outplan;
	CHECK_CUFFT_ERRORS(hipfftCreate(&outplan));
	CHECK_CUFFT_ERRORS(hipfftPlan1d(&outplan, blockSize, HIPFFT_C2R, 1));


#if defined WIN64 || CB == 0
#else
	/*Create host pointer to CB Function*/
	hipfftCallbackLoadC hostCopyOfCallbackPtr;
	checkCudaErrors(hipMemcpyFromSymbol(&hostCopyOfCallbackPtr, HIP_SYMBOL(myOwnCallbackPtr), sizeof(hostCopyOfCallbackPtr)));

	/*Associate the load callback with the plan*/
	CHECK_CUFFT_ERRORS(hipfftXtSetCallback(outplan, (void**)&hostCopyOfCallbackPtr,
		HIPFFT_CB_LD_COMPLEX, (void**)&d_filter_complex));
#endif	

	for (int i = 0; i < 4; i++) {
		checkCudaErrors(hipStreamSynchronize(stream[i]));
	}

	checkCudaErrors(hipHostFree(rbuf));

	/*Transform Filter*/
	Print("Transforming filter\n");
	CHECK_CUFFT_ERRORS(hipfftExecR2C(plan, (hipfftReal*)d_filter_complex, (hipfftComplex*)d_filter_complex));
	if (ch == 2) {
		CHECK_CUFFT_ERRORS(hipfftExecR2C(plan, (hipfftReal*)d_filter_complex + blockSize + 2, (hipfftComplex*)d_filter_complex + blockSize / 2 + 1));
	}
	/*Convolving*/
	if (flag == mono_mono) {
		Print("mono_mono Convolving\n");
		overlapAdd(d_obuf, d_filter_complex, iFrames, M, blockSize, blockNum, plan, outplan);
	}
	else if (flag == stereo_stereo) {
		Print("stereo_stereo Convolving\n");
		overlapAdd(d_obuf, d_filter_complex, iFrames, M, blockSize, blockNum, plan, outplan);
		overlapAdd(d_obuf + oFrames, d_filter_complex + blockSize / 2 + 1,
			iFrames, M, blockSize, blockNum, plan, outplan);
	}
	else if (flag == stereo_mono) {
		Print("stereo_mono Convolving\n");
		overlapAdd(d_obuf, d_filter_complex, iFrames, M, blockSize, blockNum, plan, outplan);
		overlapAdd(d_obuf + oFrames, d_filter_complex, iFrames, M, blockSize, blockNum, plan, outplan);
	}
	else {
		Print("mono_stereo Convolving\n");
		checkCudaErrors(hipMemcpy(d_obuf + oFrames, d_obuf, oFrames * sizeof(float), hipMemcpyDeviceToDevice));
		overlapAdd(d_obuf, d_filter_complex, iFrames, M, blockSize, blockNum, plan, outplan);
		overlapAdd(d_obuf + oFrames, d_filter_complex + blockSize / 2 + 1,
			iFrames, M, blockSize, blockNum, plan, outplan);
	}
	checkCudaErrors(hipFree(d_filter_complex));
	CHECK_CUFFT_ERRORS(hipfftDestroy(plan));
	CHECK_CUFFT_ERRORS(hipfftDestroy(outplan));
}

void convolutionPicker(passable* p) {
	
}
void process(passable* p) {
	float* d_ibuf = p->input->d_buf;
	float* d_rbuf = p->reverb->d_buf;
	float* d_obuf = d_ibuf;
	long long paddedSize = p->paddedSize;
	flags flag = p->type;

	/*Convolving*/
	if (flag == mono_mono) {
		Print("mono_mono Convolving\n");
		//convolve(d_ibuf, d_rbuf, paddedSize);
		convolveBatched(d_ibuf, paddedSize);
	}
	else if (flag == stereo_stereo) {
		Print("stereo_stereo Convolving\n");
		convolve(d_ibuf, d_rbuf, paddedSize);
		convolve(d_ibuf + paddedSize, d_rbuf + paddedSize, paddedSize);
	}
	else {
		mismatchedConvolve(p);
		if (flag == mono_stereo) {
			d_obuf = d_rbuf;
		}
	}
}
void asyncCopyScale(passable* p, float *obuf, long long end, float scale) {
	float* d_obuf = p->input->d_buf;
	/*Block/Thread sizes for kernels*/
	int blockSize = 512;
	int numBlocks = (end + blockSize - 1) / blockSize;

	/*Asynchronous copy & scale */
	const int nStreams = 4;
	int streamSize = (end + nStreams - 1) / nStreams;
	int streamBytes = streamSize * sizeof(float);
	numBlocks = (streamSize + blockSize - 1) / blockSize;

	/*Create streams*/
	Print("Creating streams\n");
	hipStream_t stream[nStreams];
	for (int i = 0; i < nStreams; ++i) {
		checkCudaErrors(hipStreamCreate(&stream[i]));
	}
	Print("Scaling and copying\n");
	for (int i = 0; i < nStreams; ++i) {
		long long offset = i * streamSize;
		/*Run scale kernel*/
		RealFloatScaleConcurrent << < numBlocks, blockSize, 0, stream[i] >> > (d_obuf, end, streamSize, scale, offset);
		/*Copy device memory to host asynchronously*/
		if (i == nStreams - 1) {
			streamBytes = sizeof(float) * (end - offset);
		}
		checkCudaErrors(hipMemcpyAsync(&obuf[offset], &d_obuf[offset], streamBytes, hipMemcpyDeviceToHost, stream[i]));
	}
}
float* convolutionWrapper(passable* p, bool blockProcessingOn) {
	float* d_ibuf = p->input->d_buf;
	float* d_rbuf = p->reverb->d_buf;
	float* d_obuf = d_ibuf;
	float* obuf;
	flags flag = p->type;
	int oCh = flag == mono_mono ? 1 : 2;
	long long paddedSize = p->paddedSize;
	float minmax, minmax2;
	hipEvent_t start, stop;

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

	/*Allocating host memory for output*/
	Print("Allocating host memory for output\n");
	checkCudaErrors(hipHostMalloc((void**)&obuf, paddedSize * oCh * sizeof(float)));

	/*Find peak of input signal*/
	Print("Finding peak of input signal\n");
	minmax = DExtrema(d_ibuf, paddedSize * oCh);

	/*Performing Convolution*/
	if (blockProcessingOn) {
		blockProcess(p);
	}
	else {
		process(p);
	}

	/*Find peak of output*/
	Print("Find peak of output\n");
	minmax2 = DExtrema(d_obuf, paddedSize * oCh);

	float scale = minmax / minmax2;
	long long end = paddedSize * oCh;

	asyncCopyScale(p, obuf, end, scale);

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	fprintf(stderr, "Time for GPU convolution: %f ms\n", milliseconds);

	checkCudaErrors(hipFree(d_ibuf));
	checkCudaErrors(hipFree(d_rbuf));
	return obuf;
}