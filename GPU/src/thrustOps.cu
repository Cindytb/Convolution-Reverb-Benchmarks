#include "hip/hip_runtime.h"
#include "thrustOps.cuh"

template <typename T>
struct square {
	__host__ __device__
		T operator()(const T& x) const
	{
		return x * x;
	}
};
/*Functions to find extrema*/
float DExtrema(float *pointer, long long size){
	/*Convert raw float pointer into a thrust device pointer*/
	thrust::device_ptr<float> thrust_d_signal(pointer);
	
	thrust::pair < thrust::device_ptr<float>, thrust::device_ptr<float> >blah = 
		thrust::minmax_element(thrust::device, thrust_d_signal, thrust_d_signal + size);
	float *d_min, *d_max;
	float *min = (float*)malloc(sizeof(float));
	float *max = (float*)malloc(sizeof(float));
	
	d_min = blah.first.get();
	d_max = blah.second.get();
	
	checkCudaErrors(hipMemcpy(min, d_min, sizeof(float), hipMemcpyDefault));
	checkCudaErrors(hipMemcpy(max, d_max, sizeof(float), hipMemcpyDefault));

	float result = std::abs(*min) > *max ? std::abs(*min) : *max;
	free(min);
	free(max);
	return result;
}
float DExtrema(float *pointer, long long size){
	/*Convert raw float pointer into a thrust device pointer*/
	thrust::device_ptr<float> thrust_d_signal(pointer);
	
	thrust::pair < thrust::device_ptr<float>, thrust::device_ptr<float> >blah = 
		thrust::minmax_element(thrust::device, thrust_d_signal, thrust_d_signal + size);
	float *d_min, *d_max;
	float *min = (float*)malloc(sizeof(float));
	float *max = (float*)malloc(sizeof(float));
	
	d_min = blah.first.get();
	d_max = blah.second.get();
	
	checkCudaErrors(hipMemcpy(min, d_min, sizeof(float), hipMemcpyDefault));
	checkCudaErrors(hipMemcpy(max, d_max, sizeof(float), hipMemcpyDefault));

	float result = std::abs(*min) > *max ? std::abs(*min) : *max;
	free(min);
	free(max);
	return result;
}
void fillWithZeroes(float **target_buf, long long old_size, long long new_size){
	thrust::device_ptr<float> dev_ptr(*target_buf);
	thrust::fill(dev_ptr + old_size, dev_ptr + new_size, (float) 0.0f);
}