#include "Main.cuh"

float *gpuEntry(std::string input, std::string reverb, std::string out, bool timeDomain) {
	setlocale(LC_NUMERIC, "");
	bool blockProcessingOn = false;
	/*Forward variable declarations*/
	float *obuf;
	float *buf, *rbuf;
	long long i_size = 0, r_size = 0, o_size = 0, new_size = 0;
	int iCh = 0, iSR = 0, rCh = 0, rSR = 0;

	/*Obtain audio block size based off the GPU specs*/
	long long audioBlockSize = getAudioBlockSize();
	Print("Reading file\n");
	readFile(input.c_str(), reverb.c_str(),
		&iCh, &iSR, &i_size, &rCh, &rSR, &r_size, 
		&buf, &rbuf, &new_size, &blockProcessingOn, timeDomain);
	
	o_size = i_size + r_size - 1;
	
	if(timeDomain){
		obuf = TDconvolution(&buf, &rbuf, i_size, o_size);
	}
	else{
		if(blockProcessingOn){
			int numDevs = 1;
			hipGetDeviceCount(&numDevs);
			if(numDevs == 1){
				obuf = blockConvolution(&buf, &rbuf, i_size, o_size, audioBlockSize);
			}
			else{
				obuf = multiGPUFFT(buf, rbuf, i_size, r_size);
			}
		}
		else{
			Print("Running Convolution\n");
			obuf = convolution(&buf, &rbuf, new_size, i_size, o_size);
		}
	}
	if (out.c_str()[0] != ' '){
		if (obuf != NULL){
			//fprintf(stderr, "Writing output file %s\n", out.c_str());
			writeFile(out.c_str(), obuf, o_size, iSR, iCh);
		}
	}
	
	return obuf;
}

