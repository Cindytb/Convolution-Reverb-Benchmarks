#include "hip/hip_runtime.h"
#include "Convolution.cuh"
// Define the device pointer to the callback routine. The host code will fetch this and pass it to CUFFT
__device__ hipfftCallbackLoadC myOwnCallbackPtr = cbComplexPointwiseMul;

void convolve(float **d_ibuf, float **d_rbuf, hipfftComplex **d_Cbufs, long long size){
	hipfftComplex *d_sig_complex = *d_Cbufs, *d_filter_complex = *d_Cbufs + size / 2 + 1;
	
	/*Create forward FFT plan*/
	hipfftHandle plan;
	CHECK_CUFFT_ERRORS(hipfftCreate(&plan));
	CHECK_CUFFT_ERRORS(hipfftPlan1d(&plan, size, HIPFFT_R2C, 1));

	/*Create inverse FFT plan*/
	hipfftHandle outplan;
	CHECK_CUFFT_ERRORS(hipfftCreate(&outplan));
	CHECK_CUFFT_ERRORS(hipfftPlan1d(&outplan, size, HIPFFT_C2R, 1));

#if defined WIN64 || CALLBACK == 0
	/*NO CALLBACK VERSION*/

	/*Transform Complex Signal*/
	CHECK_CUFFT_ERRORS(hipfftExecR2C(plan, (hipfftReal *) *d_ibuf, d_sig_complex));

	/*Transform Filter Signal*/
	CHECK_CUFFT_ERRORS(hipfftExecR2C(plan, (hipfftReal*) *d_rbuf, d_filter_complex));
	checkCudaErrors(hipFree(*d_rbuf));

	/*CONVOLUTION*/
	int blockSize = 256;
	int numBlocks = (size + blockSize - 1) / blockSize;
	
	ComplexPointwiseMul << < numBlocks, blockSize >> > (d_sig_complex, d_filter_complex, size / 2 + 1);
	getLastCudaError("Kernel execution failed [ ComplexPointwiseMul]");
	
	/*IFFT*/
	CHECK_CUFFT_ERRORS(hipfftExecC2R(outplan, d_sig_complex, *d_ibuf));
#else
	/*Transform Complex Signal*/
	CHECK_CUFFT_ERRORS(hipfftExecR2C(plan, (hipfftReal *)*d_ibuf, d_sig_complex));

	/*Transform Filter Signal*/
	CHECK_CUFFT_ERRORS(hipfftExecR2C(plan, (hipfftReal*)*d_rbuf, d_filter_complex));
	
	/*Copy over the host copy of callback function*/
	hipfftCallbackLoadC hostCopyOfCallbackPtr;
	checkCudaErrors(hipMemcpyFromSymbol(&hostCopyOfCallbackPtr,HIP_SYMBOL(myOwnCallbackPtr), sizeof(hostCopyOfCallbackPtr)));
	
	/*Associate the load callback with the plan*/
	CHECK_CUFFT_ERRORS(hipfftXtSetCallback(outplan, (void **)&hostCopyOfCallbackPtr, HIPFFT_CB_LD_COMPLEX, 
		(void **)&d_filter_complex));
	
	checkCudaErrors(hipFree(*d_rbuf));

	// Transform signal back, using the callback to do the pointwise multiply on the way in.
	CHECK_CUFFT_ERRORS(hipfftExecC2R(outplan, d_sig_complex, *d_ibuf));
#endif
	checkCudaErrors(hipfftDestroy(plan));
	checkCudaErrors(hipfftDestroy(outplan));
	
	checkCudaErrors(hipFree(d_sig_complex));
}

void blockConvolve(float **d_ibuf, float **d_rbuf, long long iFrames, long long rFrames){
	hipfftComplex *d_sig_complex, *d_filter_complex;
	float *d_padded_signal;
	float *d_padded_filter_kernel;
	float *d_obuf = *d_ibuf;

	int M = rFrames - 1;
	
	int myExp = ceil(log2( (float)(iFrames +M)));
	size_t blockSize = pow(2, myExp);
	int L = iFrames;
	int blockNum = 0;
	size_t workspace;
	CHECK_CUFFT_ERRORS(hipfftEstimate1d(blockSize, HIPFFT_R2C, 2, &workspace));
	while(getFreeSize() < workspace + blockSize * 18L){
		myExp--;
		blockSize = pow(2, myExp);
		blockNum++;
		CHECK_CUFFT_ERRORS(hipfftEstimate1d(blockSize, HIPFFT_R2C, 2, &workspace));
	}
	if(blockSize < iFrames + M) L = blockSize - M;
	
	/*Allocating Memory*/
	checkCudaErrors(hipMalloc(&d_filter_complex, (blockSize + 2) * sizeof(hipfftComplex)));
	checkCudaErrors(hipMalloc(&d_padded_filter_kernel, blockSize * sizeof(float)));
	d_sig_complex = d_filter_complex + blockSize / 2 + 1;	
	checkCudaErrors(hipMalloc(&d_padded_signal, blockSize * sizeof(float)));

	/*Block/Thread sizes for kernels*/
	int numThreads = 256;
	int numBlocks = (blockSize + numThreads - 1) / numThreads;
	
	/* Copy over filter */
	checkCudaErrors(hipMemcpy(d_padded_filter_kernel, *d_rbuf, rFrames * sizeof(float), hipMemcpyDeviceToDevice));
	numBlocks = (rFrames + numThreads - 1) / numThreads;
	FillWithZeros<<<numBlocks, numThreads>>>(d_padded_filter_kernel, rFrames,  blockSize);
	
	/*Free real array*/
	checkCudaErrors(hipFree(*d_rbuf));

	/*Plans*/
	hipfftHandle plan;
	CHECK_CUFFT_ERRORS(hipfftCreate(&plan));
	CHECK_CUFFT_ERRORS(hipfftPlan1d(&plan, blockSize, HIPFFT_R2C, 1));
	hipfftHandle outplan;
	CHECK_CUFFT_ERRORS(hipfftCreate(&outplan));
	CHECK_CUFFT_ERRORS(hipfftPlan1d(&outplan, blockSize, HIPFFT_C2R, 1));

	/*Transform Filter*/
	CHECK_CUFFT_ERRORS(hipfftExecR2C(plan, (hipfftReal *)d_padded_filter_kernel, d_filter_complex));
	
	/*Free real padded array*/
	checkCudaErrors(hipFree(d_padded_filter_kernel));
	
	
		
	#if defined WIN64 || CALLBACK == 0
	#else
	//fprintf(stderr, "DOING CALLBACK STUFF\n");
	/*Create host pointer to CB Function*/
	hipfftCallbackLoadC hostCopyOfCallbackPtr;
	checkCudaErrors(hipMemcpyFromSymbol(&hostCopyOfCallbackPtr,HIP_SYMBOL(myOwnCallbackPtr), sizeof(hostCopyOfCallbackPtr)));
		
	/*Associate the load callback with the plan*/
	CHECK_CUFFT_ERRORS(hipfftXtSetCallback(outplan, (void **)&hostCopyOfCallbackPtr,
		 HIPFFT_CB_LD_COMPLEX, (void **)&d_filter_complex));
	#endif	
	for(int blockNo = 0; blockNo <= blockNum; blockNo++){
		long long cpyAmount = L;
		if (blockNo == blockNum) {
			cpyAmount = iFrames % L;
		}
		//fprintf(stderr, "blockNo: %'i\tcpyAmount: %'lli\n", blockNo, cpyAmount);
		/*1/5/11/17 - Copy buf(N * L, L) -> sig[0]. cpyAmount becomes R at the end. N = 0 initially*/
		//fprintf(stderr, "Copy(block, obuf[%'i], %'i)\n", L * blockNo, cpyAmount);
		checkCudaErrors(hipMemcpy(d_padded_signal, &d_obuf[L * blockNo], cpyAmount * sizeof(float), hipMemcpyDeviceToDevice));
		if (blockNo != 0) {
			/*6/12/18 - Copy sig(L, M) -> buf[N * L]*/
			//fprintf(stderr, "Copy(obuf[%'i], block[%'i], %'i)\n", L * blockNo, L, M);
			checkCudaErrors(hipMemcpy(&d_obuf[L * blockNo], &d_padded_signal[L], M * sizeof(float), hipMemcpyDeviceToDevice));
		}
		
		/*2/7/13/19 - Pad sig(L, M) with 0's, cpyAmount becomes R at the end*/
		fillWithZeroes(&d_padded_signal, cpyAmount, blockSize);
		
		/*Transform signal*/
		CHECK_CUFFT_ERRORS(hipfftExecR2C(plan, (hipfftReal *)d_padded_signal, d_sig_complex));
		
		#if defined WIN64 || CALLBACK == 0
			//fprintf(stderr, "NO CALLBACK DOUBLE BLOCK CONVOLUTION\n");
			/*CONVOLUTION*/
			/*3/8/14/20*/
			numBlocks = (blockSize / 2 + numThreads) / numThreads;
			ComplexPointwiseMul << < numBlocks, numThreads >> > (d_sig_complex, d_filter_complex, blockSize / 2 + 1);
			getLastCudaError("Kernel execution failed [ ComplexPointwiseMul]");
		#endif
		/*IFFT*/
		CHECK_CUFFT_ERRORS(hipfftExecC2R(outplan, d_sig_complex, d_padded_signal));
		if (blockNo != 0) {
			/* 9/15/21 - Point-wise add sig(0,M) + buf[N*L]*/
			PointwiseAdd << <numBlocks, numThreads >> > (d_padded_signal, &d_obuf[blockNo * L], M);
		}
		/*Initial case*/
		if (blockNo == 0) {
			/*4 - Copy sig(0,L) -> buf[0]*/
			checkCudaErrors(hipMemcpy(d_obuf, d_padded_signal, L * sizeof(float), hipMemcpyDeviceToDevice));
		}
		/*Last case*/
		if (blockNo == blockNum) {
			//fprintf(stderr, "Copy(obuf[%'i], block[%'i], %'i)\n", blockNo * L + M, M, cpyAmount);
			checkCudaErrors(hipMemcpy(&d_obuf[blockNo * L + M], &d_padded_signal[M], cpyAmount * sizeof(float), hipMemcpyDeviceToDevice));
		}
		/*Every other case*/
		if(blockNo != 0 && blockNo < blockNum){
			/*10/16 - Copy sig(M, L-M) -> buf[N * L + M]*/
			checkCudaErrors(hipMemcpy(&d_obuf[blockNo * L + M], &d_padded_signal[M], (L - M) * sizeof(float), hipMemcpyDeviceToDevice));
		}
	}
	//Destroy CUFFT context
	CHECK_CUFFT_ERRORS(hipfftDestroy(plan));
	CHECK_CUFFT_ERRORS(hipfftDestroy(outplan));
	checkCudaErrors(hipFree(d_padded_signal));
	checkCudaErrors(hipFree(d_filter_complex));
}

float *blockConvolution(float ** d_ibuf, float ** d_rbuf, long long old_size, long long oFrames, long long audioBlockSize) {
	float *d_obuf = *d_ibuf;
	float *obuf;
	
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

	checkCudaErrors(hipHostMalloc((void**)&obuf, oFrames * sizeof(float)));
	float minmax = DExtrema(*d_ibuf, old_size);

	blockConvolve(d_ibuf, d_rbuf, old_size, oFrames - old_size + 1);

	float minmax2 = DExtrema(d_obuf, oFrames);

	float scale = minmax/minmax2;

	int blockSize = 128;
	int numBlocks = (oFrames  + blockSize - 1) / blockSize;

    int nStreams = 4;
    int streamSize = (oFrames + nStreams - 1) / nStreams;
    int streamBytes = streamSize * sizeof(float);
    hipStream_t stream[nStreams];
    for (int i = 0; i < nStreams; ++i) {
		checkCudaErrors(hipStreamCreate(&stream[i]));
    }

    /*Concurrent copy and pointwise multiply*/
    numBlocks = (streamSize + blockSize - 1) / blockSize;
    for (int i = 0; i < nStreams; ++i) {
    	int offset = i * streamSize;
    	RealFloatScaleConcurrent << < numBlocks, blockSize, 0, stream[i] >> > (d_obuf, oFrames, streamSize, scale, offset);
    	
    	if ( i == nStreams - 1){
    		 streamBytes = (oFrames - offset) * sizeof(float);
    	}
		checkCudaErrors(hipMemcpyAsync(&obuf[offset], &d_obuf[offset], streamBytes, hipMemcpyDeviceToHost, stream[i]));
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	fprintf(stderr,"Time for GPU convolution: %f ms\n", milliseconds);
	checkCudaErrors(hipFree(*d_ibuf));

	return obuf;

}
/*Convolution with device memory allocated previously*/
float *convolution(float **d_ibuf, float ** d_rbuf, long long size, long long old_size, long long oFrames) {
	hipfftComplex *d_complex;
	float *d_obuf = *d_ibuf;
	float *obuf;
	
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	/*Allocate memory for complex signal & filter*/
	checkCudaErrors(hipMalloc(&d_complex, (size + 2)* sizeof(hipfftComplex)));
	
	checkCudaErrors(hipHostMalloc((void**)&obuf, size * sizeof(float)));
	
	/*Find peak of input signal*/
	float minmax = DExtrema(*d_ibuf, old_size);

	/*Convolving*/
	convolve(d_ibuf, d_rbuf, &d_complex, size);

	/*Find peak of output*/
	float minmax2 = DExtrema(d_obuf, size);
	float scale = minmax/minmax2;
	
	
	/*Block/Thread sizes for kernels*/
	int strides = 1;
	int blockSize = 128;
	int numBlocks = (size + blockSize - 1) / blockSize;
	numBlocks = ( oFrames / strides + blockSize - 1) / blockSize;
	while (numBlocks > (2U << 31 - 1)) {
		numBlocks = ( oFrames  / ++strides + blockSize - 1) / blockSize;
	}
	
	/*Asynchronous copy & scale */
	int nStreams = 4;
	//printf("number of streams: %'i\n", nStreams);
	int streamSize = (oFrames + nStreams - 1) / nStreams;
	int streamBytes = streamSize * sizeof(float);

	hipStream_t stream[nStreams];
	for (int i = 0; i < nStreams; ++i) {
		checkCudaErrors(hipStreamCreate(&stream[i]));
	}
	/*Scale resulting signal according to input signal*/
	numBlocks = (streamSize + blockSize - 1) / blockSize;
	for (int i = 0; i < nStreams; ++i) {
		int offset = i * streamSize;
		/*Run scale kernel*/
		RealFloatScaleConcurrent << < numBlocks, blockSize, 0, stream[i] >> > (d_obuf, size, streamSize, scale, offset);
		/*Copy device memory to host asynchronously*/
		if(i == nStreams - 1) streamBytes = sizeof(float) * (oFrames - offset);
		checkCudaErrors(hipMemcpyAsync(&obuf[offset], &d_obuf[offset], streamBytes, hipMemcpyDeviceToHost, stream[i]));
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	fprintf(stderr,"Time for GPU convolution: %f ms\n", milliseconds);

	checkCudaErrors(hipFree(d_obuf));
	return obuf;
}
float *multiGPUFFT(float *ibuf, float *rbuf, long long iFrames, long long rFrames) {
	setlocale(LC_NUMERIC, "");
	long long oFrames = iFrames + rFrames - 1;
	int streamsPerDev = 4;
	
	/*get number of devices*/
	int numDevs = 0;
	hipGetDeviceCount(&numDevs);
	hipStream_t stream[numDevs * streamsPerDev];
	/*Establish all arrays for number of devices*/
	float *d_ibufs[numDevs], *d_rbufs[numDevs];
	hipfftComplex *d_Cbufs[numDevs];
	float *d_obuf, *obuf;
	size_t inSizes[numDevs];
	bool doubleBlock = false;
	int amtPerDevice, M = rFrames - 1;
	int singleDev = 0;
	size_t maxFree = 0;
	size_t freeSizes[numDevs];

	Print("Finding free memory on each device\n");
	/*Find out amount of free memory on each device*/
	
	for (int i = 0; i < numDevs; i++) {
		hipSetDevice(i);
		freeSizes[i] = getFreeSize();
		/*most precise is input = freeSize()/16 - 16, but dividing by 32 to conservatively account for cuFFT space*/
		size_t freeSize = freeSizes[i] / 32;
		/*max number of elements that's a power of 2*/
		inSizes[i] = pow(2, floor(log2((double)freeSize)));	
		//fprintf(stderr, "inSizes[%i] = %lli\n", i, inSizes[i]);
	}

	long long totalAllowedFrames = 0;
	for(int i = 0; i < numDevs; i++){
		totalAllowedFrames += inSizes[i] - M;
	}
	
	long long frames = 0;
	/*Allocating memory for normal case*/
	if (totalAllowedFrames > iFrames){
		Print("Allocating memory single block\n");
		for(int i = 0; i < numDevs; i++){
			if(frames >= iFrames) {
				inSizes[i] = 0;
				continue;
			}
			hipSetDevice(i);
			frames += inSizes[i] - M;
			//fprintf(stderr, "Allocating device %i: blockSize: %i\n", i, inSizes[i]);
			checkCudaErrors(hipMalloc(&d_ibufs[i], inSizes[i] * sizeof(float)));
			checkCudaErrors(hipMalloc(&d_rbufs[i], inSizes[i] * sizeof(float)));
			checkCudaErrors(hipMalloc(&d_Cbufs[i], (inSizes[i] + 2) * sizeof(hipfftComplex)));
		}
	}
	
	else{
		Print("Verifying total size of GPUs\n");
		totalAllowedFrames = 0;
		for(int i = 0; i < numDevs; i++){
			/*Theoretically should be 4. Dividing by 8 to be conservative*/
			totalAllowedFrames += freeSizes[i] / 4;
			totalAllowedFrames -= rFrames;
			if (maxFree < freeSizes[i]){
				maxFree = freeSizes[i];
				singleDev = i;
			}
		}
		if(totalAllowedFrames < iFrames + M * numDevs || 
			freeSizes[singleDev] / 4 - inSizes[singleDev] < oFrames + M){
			fprintf(stderr, "\n\nERROR: NOT ENOUGH COLLECTIVE MEMORY ON THE GPUs. EXITING\n\n");
			checkCudaErrors(hipHostFree(ibuf));
			free(rbuf);
			return NULL;
		}
		Print("Allocating memory double block\n");
		/*Allocating memory for double block case*/
		amtPerDevice = (iFrames + numDevs - 1) / numDevs;
		for(int i = 0; i < numDevs; i++){
			size_t freeSize = freeSizes[i] / 8 - rFrames;
			freeSize = pow(2, floor(log2((double)freeSize)));
			if (amtPerDevice + M > freeSize){
				fprintf(stderr, "WARNING: One GPU doesn't have enough memory. Redistributing memory.\n");
				amtPerDevice = iFrames;
				break;
			}
		}
		
		long long framecount = 0;
		doubleBlock = true;
		for(int i = 0; i < numDevs; i++){
			hipSetDevice(i);
			/*Theoretically should be 4. Dividing by 8 to be conservative*/
			size_t freeSize = freeSizes[i] / 8;
			freeSize -= rFrames;
			freeSize = pow(2, floor(log2((double)freeSize)));
			int currFrames = amtPerDevice;
			if (currFrames + M > freeSize){
				currFrames = freeSize - M;
			}
			
			if(framecount + currFrames > iFrames){
				currFrames = iFrames - framecount;
			}
			
			if(currFrames == 0){
				inSizes[i] = 0;
				continue;
			}
			inSizes[i] = currFrames + M;
			checkCudaErrors(hipMalloc(&d_ibufs[i], inSizes[i] * sizeof(float)));
			checkCudaErrors(hipMalloc(&d_rbufs[i], rFrames * sizeof(float)));
			framecount += currFrames;
		}
		if(framecount < iFrames){
			fprintf(stderr, "\n\nERROR: NOT ENOUGH COLLECTIVE MEMORY ON THE GPUs. EXITING\n\n");
			checkCudaErrors(hipHostFree(ibuf));
			for(int i = 0; i < numDevs; i++){
				hipSetDevice(i);
				checkCudaErrors(hipFree(d_ibufs[i]));
				checkCudaErrors(hipFree(d_rbufs[i]));
			}
			free(rbuf);
			return NULL;
		}
	}
	
	/**
	{
	TODO: Peer-to-Peer memcpy of rbuf
	hipDeviceProp_t prop;
	for(int i = 0; i )
	checkCudaErrors(hipGetDeviceProperties(&prop, ))
	int rbufDevNum = 0;
	for(int i = 0; i < numDevs; i++){
		for(int j = 0; j < numDevs; j++){
			if (i == j) continue;
			int num = 0;
			hipDeviceCanAccessPeer(&num, i, j);
			if(num){
				hipMemcpyAsync(d_rbufs[i], rbuf, rFrames * sizeof(float));
				rbufDevNum = i;
			}
		}
	}
	**/
	long long blockSize = 512;
	int numBlocks;
	/*Copy each chunk of input into each GPU and pad with 0's*/
	frames = 0;
	Print("Copying memory\n");
	for(int i = 0; i < numDevs; i++){
		hipSetDevice(i);
		checkCudaErrors(hipStreamCreate(&stream[i * streamsPerDev]));
		checkCudaErrors(hipStreamCreate(&stream[i * streamsPerDev + 1]));
		checkCudaErrors(hipStreamCreate(&stream[i * streamsPerDev + 2]));
		checkCudaErrors(hipStreamCreate(&stream[i * streamsPerDev + 3]));
		if (inSizes[i] == 0){
			continue;
		}
		long long amtRead = inSizes[i] - M;
		if (frames + amtRead > iFrames){
			amtRead = iFrames - frames;
		}
				
		checkCudaErrors(hipMemcpyAsync(d_ibufs[i], ibuf + frames, amtRead * sizeof(float), 
			hipMemcpyHostToDevice, stream[i * streamsPerDev]));
	
		numBlocks = (inSizes[i] - amtRead + blockSize - 1 ) / blockSize;
		FillWithZeros<<<numBlocks, blockSize, 0, stream[i * streamsPerDev + 1]>>>(d_ibufs[i], amtRead, inSizes[i]);
		if(!doubleBlock){	
			numBlocks = (inSizes[i] - rFrames - 1 + blockSize) / blockSize;
			FillWithZeros<<<numBlocks, blockSize, 0, stream[i * streamsPerDev + 2]>>>(d_rbufs[i], rFrames, inSizes[i]);
		}
		/*WILL BE REPLACED LATER*/
		checkCudaErrors(hipMemcpyAsync(d_rbufs[i], rbuf, rFrames * sizeof(float), hipMemcpyHostToDevice, stream[i * streamsPerDev + 3]));
		//fprintf(stderr, "Copying reverb\n");
		//checkCudaErrors(hipMemcpy(d_rbufs[i], rbuf, rFrames * sizeof(float), hipMemcpyHostToDevice));
		///////////////////////////////////////////////////////////
		frames += amtRead;
	}
	
	checkCudaErrors(hipSetDevice(0));
	hipEvent_t start, stop;
	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));
	checkCudaErrors(hipEventRecord(start));
	/*Loop through all input buffers and find the peak*/
	frames = iFrames;
	float minmax1 = 0;
	Print("Find Overall peak\n");
	for(int i = 0 ; i < numDevs; i++){
		checkCudaErrors(hipSetDevice(i));
		checkCudaErrors(hipStreamSynchronize(stream[i * streamsPerDev]));
		checkCudaErrors(hipStreamSynchronize(stream[i * streamsPerDev + 1]));
		if(inSizes[i] == 0) break;
		frames -= inSizes[i] - M;
		float minmax = DExtrema(d_ibufs[i], inSizes[i]);
		if(minmax > minmax1)
			minmax1 = minmax;
	}
	checkCudaErrors(hipHostFree(ibuf));
	for(int i = 0; i < numDevs; i++){
		checkCudaErrors(hipSetDevice(i));
		checkCudaErrors(hipStreamSynchronize(stream[i * streamsPerDev + 3]));
	}
	free(rbuf);
	/*Convolve all chunks*/
	frames = iFrames;

	if(doubleBlock){
		Print("Double Block Convolution\n");
		for(int i = 0; i < numDevs; i++){
			hipSetDevice(i);
			checkCudaErrors(hipStreamSynchronize(stream[i * streamsPerDev + 1]));
			checkCudaErrors(hipStreamSynchronize(stream[i * streamsPerDev + 2]));
			checkCudaErrors(hipStreamSynchronize(stream[i * streamsPerDev + 3]));
			if(inSizes[i] == 0) break;
			blockConvolve(&d_ibufs[i], &d_rbufs[i], inSizes[i] - M, rFrames);
		}
	}
	else{
		Print("Single Block Convolution\n");
		for(int i = 0; i < numDevs; i++){
			hipSetDevice(i);
			checkCudaErrors(hipStreamSynchronize(stream[i * streamsPerDev + 2]));
			if(inSizes[i] == 0) break;
			convolve(&d_ibufs[i], &d_rbufs[i], &d_Cbufs[i], inSizes[i]);
		}
	}


	/*Overlap-add method to combine the convolved chunks*/
	Print("Overlap-add Reconstruction\n");
	maxFree = 0;
	for(int i = 0; i < numDevs; i++){
		checkCudaErrors(hipSetDevice(i));
		size_t size = getFreeSize();
		if(maxFree > size){
			maxFree = size;
			singleDev = i;
		}
	}
	checkCudaErrors(hipSetDevice(singleDev));
	//printSize();
	float *d_scratchSpace;
	checkCudaErrors(hipHostMalloc(&obuf, oFrames * sizeof(float)));
	checkCudaErrors(hipMalloc(&d_obuf, oFrames * sizeof(float)));
	checkCudaErrors(hipMemcpyAsync(d_obuf, d_ibufs[0], inSizes[0] * sizeof(float), hipMemcpyDefault, stream[singleDev * streamsPerDev]));
	checkCudaErrors(hipMalloc(&d_scratchSpace, M * sizeof(float)));
	checkCudaErrors(hipStreamSynchronize(stream[singleDev * streamsPerDev]));
	hipSetDevice(0);
	checkCudaErrors(hipFree(d_ibufs[0]));
	
	long long size = inSizes[0];
	for(int i = 1; i < numDevs; i++){
		long long cpyAmount = inSizes[i] - M;
		if (size + cpyAmount > iFrames) {
			cpyAmount = oFrames - size;
		}
		if(inSizes[i] == 0) break;
		hipSetDevice(i);
		checkCudaErrors(hipMemcpyAsync(d_scratchSpace, d_ibufs[i], M * sizeof(float), 
			hipMemcpyDefault, stream[i * streamsPerDev + 1]));
		checkCudaErrors(hipMemcpyAsync(d_obuf + size, d_ibufs[i] + M , cpyAmount * sizeof(float), 
			hipMemcpyDefault, stream[i * streamsPerDev]));
		checkCudaErrors(hipStreamSynchronize(stream[i * streamsPerDev + 1]));
		hipSetDevice(singleDev);
		numBlocks = (M + blockSize - 1) / blockSize;
		PointwiseAdd <<< numBlocks, blockSize, 0, stream[singleDev * streamsPerDev] >>>(d_scratchSpace, d_obuf + size - M, M);
		
		size += inSizes[i] - M;
		if(size >= oFrames){
			break;
		}	
	}
	frames = iFrames;
	for(int i = 0; i < numDevs; i++){
		frames -= inSizes[i] - M;
		checkCudaErrors(hipSetDevice(i));
		checkCudaErrors(hipStreamSynchronize(stream[i * 4]));
		checkCudaErrors(hipStreamSynchronize(stream[i * 4 + 1]));
		checkCudaErrors(hipStreamSynchronize(stream[i * 4 + 2]));
		checkCudaErrors(hipStreamSynchronize(stream[i * 4 + 3]));

		if(i != singleDev){
			checkCudaErrors(hipStreamDestroy(stream[i * 4]));
			checkCudaErrors(hipStreamDestroy(stream[i * 4 + 1]));
			checkCudaErrors(hipStreamDestroy(stream[i * 4 + 2]));
			checkCudaErrors(hipStreamDestroy(stream[i * 4 + 3]));
		}
		else{
			checkCudaErrors(hipFree(d_scratchSpace));
		}
		if( i != 0 && inSizes[i] != 0)
			checkCudaErrors(hipFree(d_ibufs[i]));
	}
	Print("Attempting to find output peak\n");
	checkCudaErrors(hipSetDevice(singleDev));
	checkCudaErrors(hipStreamSynchronize(stream[singleDev * 4]));
	checkCudaErrors(hipStreamSynchronize(stream[singleDev * 4 + 1]));
	checkCudaErrors(hipStreamSynchronize(stream[singleDev * 4 + 2]));
	checkCudaErrors(hipStreamSynchronize(stream[singleDev * 4 + 3]));
	float minmax2;
	Print("Finding extrema\n");
	minmax2 = DExtrema(d_obuf, oFrames);
	float scale = minmax1/minmax2;
	
	Print("Scaling and copying out\n");
	int strides = 1;
	blockSize = 128;
	numBlocks = (oFrames / strides + blockSize - 1) / blockSize;
	while (numBlocks >(2U << 31 - 1)) {
		numBlocks = (oFrames / ++strides + blockSize - 1) / blockSize;
	}
	
	
	int nStreams = 4;
	int streamSize = (oFrames + nStreams - 1) / nStreams;
	int streamBytes = streamSize * sizeof(float);

	
	/*Scale + copy 4x*/
	numBlocks = (streamSize + blockSize - 1) / blockSize;
	for (int i = 0; i < nStreams; ++i) {
		int offset = i * streamSize;
		RealFloatScaleConcurrent << < numBlocks, blockSize, 0, stream[singleDev * streamsPerDev + i] >> > (d_obuf, oFrames, streamSize, scale, offset);
		if ( i == nStreams - 1){
			streamBytes = (oFrames - offset) * sizeof(float);
		}
		checkCudaErrors(hipMemcpyAsync(&obuf[offset], &d_obuf[offset], streamBytes, hipMemcpyDeviceToHost, stream[singleDev * streamsPerDev + i]));
	}
	for(int i = 0; i < 4; i++){
		checkCudaErrors(hipStreamSynchronize(stream[singleDev  * streamsPerDev + i]));
		checkCudaErrors(hipStreamDestroy(stream[singleDev  * streamsPerDev + i]));
	}
	checkCudaErrors(hipSetDevice(0));
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	fprintf(stderr,"Time for GPU convolution: %f ms\n", milliseconds);
	checkCudaErrors(hipFree(d_obuf));
	return obuf;

}
