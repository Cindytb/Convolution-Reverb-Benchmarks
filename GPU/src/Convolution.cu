#include "hip/hip_runtime.h"
#include "Convolution.cuh"
// Define the device pointer to the callback routine. The host code will fetch this and pass it to CUFFT
__device__ hipfftCallbackLoadC myOwnCallbackPtr = cbComplexPointwiseMul;

void convolve(float **d_ibuf, float **d_rbuf, hipfftComplex **d_Cbufs, long long size){
	hipfftComplex *d_sig_complex = *d_Cbufs, *d_filter_complex = *d_Cbufs + size / 2 + 1;
	
	/*Create forward FFT plan*/
	hipfftHandle plan;
	CHECK_CUFFT_ERRORS(hipfftCreate(&plan));
	CHECK_CUFFT_ERRORS(hipfftPlan1d(&plan, size, HIPFFT_R2C, 1));

	/*Create inverse FFT plan*/
	hipfftHandle outplan;
	CHECK_CUFFT_ERRORS(hipfftCreate(&outplan));
	CHECK_CUFFT_ERRORS(hipfftPlan1d(&outplan, size, HIPFFT_C2R, 1));

#if defined WIN64 || CALLBACK == 0
	/*NO CALLBACK VERSION*/

	/*Transform Complex Signal*/
	CHECK_CUFFT_ERRORS(hipfftExecR2C(plan, (hipfftReal *) *d_ibuf, d_sig_complex));

	/*Transform Filter Signal*/
	CHECK_CUFFT_ERRORS(hipfftExecR2C(plan, (hipfftReal*) *d_rbuf, d_filter_complex));
	checkCudaErrors(hipFree(*d_rbuf));

	/*CONVOLUTION*/
	int blockSize = 256;
	int numBlocks = (size + blockSize - 1) / blockSize;
	
	ComplexPointwiseMul << < numBlocks, blockSize >> > (d_sig_complex, d_filter_complex, size / 2 + 1);
	getLastCudaError("Kernel execution failed [ ComplexPointwiseMul]");
	
	/*IFFT*/
	CHECK_CUFFT_ERRORS(hipfftExecC2R(outplan, d_sig_complex, *d_ibuf));
#else
	/*Transform Complex Signal*/
	CHECK_CUFFT_ERRORS(hipfftExecR2C(plan, (hipfftReal *)*d_ibuf, d_sig_complex));

	/*Transform Filter Signal*/
	CHECK_CUFFT_ERRORS(hipfftExecR2C(plan, (hipfftReal*)*d_rbuf, d_filter_complex));
	
	/*Copy over the host copy of callback function*/
	hipfftCallbackLoadC hostCopyOfCallbackPtr;
	checkCudaErrors(hipMemcpyFromSymbol(&hostCopyOfCallbackPtr,HIP_SYMBOL(myOwnCallbackPtr), sizeof(hostCopyOfCallbackPtr)));
	
	/*Associate the load callback with the plan*/
	CHECK_CUFFT_ERRORS(hipfftXtSetCallback(outplan, (void **)&hostCopyOfCallbackPtr, HIPFFT_CB_LD_COMPLEX, 
		(void **)&d_filter_complex));
	
	checkCudaErrors(hipFree(*d_rbuf));

	// Transform signal back, using the callback to do the pointwise multiply on the way in.
	CHECK_CUFFT_ERRORS(hipfftExecC2R(outplan, d_sig_complex, *d_ibuf));
#endif
	checkCudaErrors(hipfftDestroy(plan));
	checkCudaErrors(hipfftDestroy(outplan));
	
	checkCudaErrors(hipFree(d_sig_complex));
}

void blockConvolve(float **d_ibuf, float **d_rbuf, long long iFrames, long long rFrames){
	hipfftComplex *d_sig_complex, *d_filter_complex;
	float *d_padded_signal;
	float *d_padded_filter_kernel;
	float *d_obuf = *d_ibuf;

	int M = rFrames - 1;
	
	int myExp = ceil(log2( (float)(iFrames +M)));
	size_t blockSize = pow(2, myExp);
	int L = iFrames;
	int blockNum = 0;
	size_t workspace;
	CHECK_CUFFT_ERRORS(hipfftEstimate1d(blockSize, HIPFFT_R2C, 2, &workspace));
	while(getFreeSize() < workspace + blockSize * 18L){
		myExp--;
		blockSize = pow(2, myExp);
		blockNum++;
		CHECK_CUFFT_ERRORS(hipfftEstimate1d(blockSize, HIPFFT_R2C, 2, &workspace));
	}
	if(blockSize < iFrames + M) L = blockSize - M;
	
	/*Allocating Memory*/
	checkCudaErrors(hipMalloc(&d_filter_complex, (blockSize + 2) * sizeof(hipfftComplex)));
	checkCudaErrors(hipMalloc(&d_padded_filter_kernel, blockSize * sizeof(float)));
	d_sig_complex = d_filter_complex + blockSize / 2 + 1;	
	checkCudaErrors(hipMalloc(&d_padded_signal, blockSize * sizeof(float)));

	/*Block/Thread sizes for kernels*/
	int numThreads = 256;
	int numBlocks = (blockSize + numThreads - 1) / numThreads;
	
	/* Copy over filter */
	checkCudaErrors(hipMemcpy(d_padded_filter_kernel, *d_rbuf, rFrames * sizeof(float), hipMemcpyDeviceToDevice));
	numBlocks = (rFrames + numThreads - 1) / numThreads;
	FillWithZeros<<<numBlocks, numThreads>>>(d_padded_filter_kernel, rFrames,  blockSize);
	
	/*Free real array*/
	checkCudaErrors(hipFree(*d_rbuf));

	/*Plans*/
	hipfftHandle plan;
	CHECK_CUFFT_ERRORS(hipfftCreate(&plan));
	CHECK_CUFFT_ERRORS(hipfftPlan1d(&plan, blockSize, HIPFFT_R2C, 1));
	hipfftHandle outplan;
	CHECK_CUFFT_ERRORS(hipfftCreate(&outplan));
	CHECK_CUFFT_ERRORS(hipfftPlan1d(&outplan, blockSize, HIPFFT_C2R, 1));

	/*Transform Filter*/
	CHECK_CUFFT_ERRORS(hipfftExecR2C(plan, (hipfftReal *)d_padded_filter_kernel, d_filter_complex));
	
	/*Free real padded array*/
	checkCudaErrors(hipFree(d_padded_filter_kernel));
	
	
		
	#if defined WIN64 || CALLBACK == 0
	#else
	//fprintf(stderr, "DOING CALLBACK STUFF\n");
	/*Create host pointer to CB Function*/
	hipfftCallbackLoadC hostCopyOfCallbackPtr;
	checkCudaErrors(hipMemcpyFromSymbol(&hostCopyOfCallbackPtr,HIP_SYMBOL(myOwnCallbackPtr), sizeof(hostCopyOfCallbackPtr)));
		
	/*Associate the load callback with the plan*/
	CHECK_CUFFT_ERRORS(hipfftXtSetCallback(outplan, (void **)&hostCopyOfCallbackPtr,
		 HIPFFT_CB_LD_COMPLEX, (void **)&d_filter_complex));
	#endif	
	for(int blockNo = 0; blockNo <= blockNum; blockNo++){
		long long cpyAmount = L;
		if (blockNo == blockNum) {
			cpyAmount = iFrames % L;
		}
		//fprintf(stderr, "blockNo: %'i\tcpyAmount: %'lli\n", blockNo, cpyAmount);
		/*1/5/11/17 - Copy buf(N * L, L) -> sig[0]. cpyAmount becomes R at the end. N = 0 initially*/
		//fprintf(stderr, "Copy(block, obuf[%'i], %'i)\n", L * blockNo, cpyAmount);
		checkCudaErrors(hipMemcpy(d_padded_signal, &d_obuf[L * blockNo], cpyAmount * sizeof(float), hipMemcpyDeviceToDevice));
		if (blockNo != 0) {
			/*6/12/18 - Copy sig(L, M) -> buf[N * L]*/
			//fprintf(stderr, "Copy(obuf[%'i], block[%'i], %'i)\n", L * blockNo, L, M);
			checkCudaErrors(hipMemcpy(&d_obuf[L * blockNo], &d_padded_signal[L], M * sizeof(float), hipMemcpyDeviceToDevice));
		}
		
		/*2/7/13/19 - Pad sig(L, M) with 0's, cpyAmount becomes R at the end*/
		
		/*2/7/13/19 - Pad sig(L, M) with 0's, cpyAmount becomes R at the end*/
		//fprintf(stderr, "padZeroes(block, %'i, %'i)\n", cpyAmount, blockSize);
		fillWithZeroes(&d_padded_signal, cpyAmount, blockSize);
		//numBlocks = (blockSize - cpyAmount + numThreads - 1) / numThreads;
		//FillWithZeros<<<numBlocks, numThreads>>>(d_padded_signal, cpyAmount, blockSize);
		
		/*Transform signal*/
		//fprintf(stderr, "FFT block\n");
		CHECK_CUFFT_ERRORS(hipfftExecR2C(plan, (hipfftReal *)d_padded_signal, d_sig_complex));
		
		#if defined WIN64 || CALLBACK == 0
			//fprintf(stderr, "NO CALLBACK DOUBLE BLOCK CONVOLUTION\n");
			/*CONVOLUTION*/
			/*3/8/14/20*/
			numBlocks = (blockSize / 2 + numThreads) / numThreads;
			ComplexPointwiseMul << < numBlocks, numThreads >> > (d_sig_complex, d_filter_complex, blockSize / 2 + 1);
			getLastCudaError("Kernel execution failed [ ComplexPointwiseMul]");
		#endif
		/*IFFT*/
		//fprintf(stderr, "Pointwise multiply & IFFT Block\n");
		CHECK_CUFFT_ERRORS(hipfftExecC2R(outplan, d_sig_complex, d_padded_signal));
		if (blockNo != 0) {
			/* 9/15/21 - Point-wise add sig(0,M) + buf[N*L]*/
			//fprintf(stderr, "Add(obuf, block[%'i], %'i)\n", blockNo * L, M);
			PointwiseAdd << <numBlocks, numThreads >> > (d_padded_signal, &d_obuf[blockNo * L], M);
		}
		/*Initial case*/
		if (blockNo == 0) {
			/*4 - Copy sig(0,L) -> buf[0]*/
			//fprintf(stderr, "Copy(obuf, block, %'i)\n", L);
			checkCudaErrors(hipMemcpy(d_obuf, d_padded_signal, L * sizeof(float), hipMemcpyDeviceToDevice));
		}
		/*Last case*/
		if (blockNo == blockNum) {
			//fprintf(stderr, "Copy(obuf[%'i], block[%'i], %'i)\n", blockNo * L + M, M, cpyAmount);
			checkCudaErrors(hipMemcpy(&d_obuf[blockNo * L + M], &d_padded_signal[M], cpyAmount * sizeof(float), hipMemcpyDeviceToDevice));
		}
		/*Every other case*/
		if(blockNo != 0 && blockNo < blockNum){
			/*10/16 - Copy sig(M, L-M) -> buf[N * L + M]*/
			//fprintf(stderr, "Copy(obuf[%'i], block[%'i], %'i)\n", blockNo * L + M, M, L - M);
			checkCudaErrors(hipMemcpy(&d_obuf[blockNo * L + M], &d_padded_signal[M], (L - M) * sizeof(float), hipMemcpyDeviceToDevice));
		}
	}
	//Destroy CUFFT context
	CHECK_CUFFT_ERRORS(hipfftDestroy(plan));
	CHECK_CUFFT_ERRORS(hipfftDestroy(outplan));
	checkCudaErrors(hipFree(d_padded_signal));
	checkCudaErrors(hipFree(d_filter_complex));
}

float *blockConvolution(float ** d_ibuf, float ** d_rbuf, long long old_size, long long oFrames, long long audioBlockSize) {
	float *d_obuf = *d_ibuf;
	float *obuf;
	
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

	checkCudaErrors(hipHostMalloc((void**)&obuf, oFrames * sizeof(float)));
	float minmax = DExtrema(*d_ibuf, old_size);

	blockConvolve(d_ibuf, d_rbuf, old_size, oFrames - old_size + 1);

	float minmax2 = DExtrema(d_obuf, oFrames);

	float scale = minmax/minmax2;

	int blockSize = 128;
	int numBlocks = (oFrames  + blockSize - 1) / blockSize;

    int nStreams = 4;
    int streamSize = (oFrames + nStreams - 1) / nStreams;
    int streamBytes = streamSize * sizeof(float);
    hipStream_t stream[nStreams];
    for (int i = 0; i < nStreams; ++i) {
		checkCudaErrors(hipStreamCreate(&stream[i]));
    }

    /*Concurrent copy and pointwise multiply*/
    numBlocks = (streamSize + blockSize - 1) / blockSize;
    for (int i = 0; i < nStreams; ++i) {
    	int offset = i * streamSize;
    	RealFloatScaleConcurrent << < numBlocks, blockSize, 0, stream[i] >> > (d_obuf, oFrames, streamSize, scale, offset);
    	
    	if ( i == nStreams - 1){
    		 streamBytes = (oFrames - offset) * sizeof(float);
    	}
		checkCudaErrors(hipMemcpyAsync(&obuf[offset], &d_obuf[offset], streamBytes, hipMemcpyDeviceToHost, stream[i]));
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	fprintf(stderr,"Time for GPU convolution: %f ms\n", milliseconds);
	checkCudaErrors(hipFree(*d_ibuf));

	return obuf;

}
/*Convolution with device memory allocated previously*/
float *convolution(float **d_ibuf, float ** d_rbuf, long long size, long long old_size, long long oFrames) {
	hipfftComplex *d_complex;
	float *d_obuf = *d_ibuf;
	float *obuf;
	
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	/*Allocate memory for complex signal & filter*/
	checkCudaErrors(hipMalloc(&d_complex, (size + 2)* sizeof(hipfftComplex)));
	
	checkCudaErrors(hipHostMalloc((void**)&obuf, size * sizeof(float)));
	
	/*Find peak of input signal*/
	float minmax = DExtrema(*d_ibuf, old_size);

	/*Convolving*/
	convolve(d_ibuf, d_rbuf, &d_complex, size);

	/*Find peak of output*/
	float minmax2 = DExtrema(d_obuf, size);
	float scale = minmax/minmax2;
	
	
	/*Block/Thread sizes for kernels*/
	int strides = 1;
	int blockSize = 128;
	int numBlocks = (size + blockSize - 1) / blockSize;
	numBlocks = ( oFrames / strides + blockSize - 1) / blockSize;
	while (numBlocks > (2U << 31 - 1)) {
		numBlocks = ( oFrames  / ++strides + blockSize - 1) / blockSize;
	}
	
	/*Asynchronous copy & scale */
	int nStreams = 4;
	//printf("number of streams: %'i\n", nStreams);
	int streamSize = (oFrames + nStreams - 1) / nStreams;
	int streamBytes = streamSize * sizeof(float);

	hipStream_t stream[nStreams];
	for (int i = 0; i < nStreams; ++i) {
		checkCudaErrors(hipStreamCreate(&stream[i]));
	}
	/*Scale resulting signal according to input signal*/
	numBlocks = (streamSize + blockSize - 1) / blockSize;
	for (int i = 0; i < nStreams; ++i) {
		int offset = i * streamSize;
		/*Run scale kernel*/
		RealFloatScaleConcurrent << < numBlocks, blockSize, 0, stream[i] >> > (d_obuf, size, streamSize, scale, offset);
		/*Copy device memory to host asynchronously*/
		if(i == nStreams - 1) streamBytes = sizeof(float) * (oFrames - offset);
		checkCudaErrors(hipMemcpyAsync(&obuf[offset], &d_obuf[offset], streamBytes, hipMemcpyDeviceToHost, stream[i]));
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	fprintf(stderr,"Time for GPU convolution: %f ms\n", milliseconds);

	checkCudaErrors(hipFree(d_obuf));
	return obuf;
}
float *multiGPUFFT(float *ibuf, float *rbuf, long long iFrames, long long rFrames) {
	setlocale(LC_NUMERIC, "");
	long long oFrames = iFrames + rFrames - 1;

	/*get number of devices*/
	int numDevs = 0;
	hipGetDeviceCount(&numDevs);

	/*Establish all arrays for number of devices*/
	float *d_ibufs[numDevs], *d_rbufs[numDevs];
	hipfftComplex *d_Cbufs[numDevs];
	float *d_obuf, *obuf;
	size_t inSizes[numDevs];
	bool doubleBlock = false;
	int amtPerDevice, M = rFrames - 1;
	if( (size_t) oFrames * (size_t)6 > getFreeSize()){
		fprintf(stderr, "ERROR: Device 0 does not have enough memory for thrust operation. Exiting program\n");
		checkCudaErrors(hipHostFree(ibuf));
		free(rbuf);
		return NULL;
	}

	/*Find out amount of free memory on each device*/
	long long frames = 0;
	for (int i = 0; i < numDevs; i++) {
		hipSetDevice(i);
		//most precise is input = freeSize()/16 - 16, but dividing by 32 to conservatively account for cuFFT space
		size_t freeSize = getFreeSize() / 32;
		/*max number of elements that's a power of 2*/
		inSizes[i] = pow(2, floor(log2((double)freeSize)));
		
		
	}

	long long totalAllowedFrames = 0;
	for(int i = 0; i < numDevs; i++){
		totalAllowedFrames += inSizes[i] - M;
	}
	/*Allocating memory for normal case*/
	if (totalAllowedFrames > iFrames){
		for(int i = 0; i < numDevs; i++){
			hipSetDevice(i);
			if(frames >= iFrames) break;
			frames += inSizes[i] - M;
			checkCudaErrors(hipMalloc(&d_ibufs[i], inSizes[i] * sizeof(float)));
			checkCudaErrors(hipMalloc(&d_rbufs[i], inSizes[i] * sizeof(float)));
			checkCudaErrors(hipMalloc(&d_Cbufs[i], (inSizes[i] + 2) * sizeof(hipfftComplex)));
		}
	}
	
	else{

		totalAllowedFrames = 0;
		for(int i = 0; i < numDevs; i++){
			hipSetDevice(i);
			totalAllowedFrames += getFreeSize() / 4;
			totalAllowedFrames -= rFrames;
		}
		if(totalAllowedFrames < iFrames + M * numDevs){
			fprintf(stderr, "\n\nERROR: NOT ENOUGH COLLETIVE MEMORY ON THE GPUs. EXITING\n\n");
			checkCudaErrors(hipHostFree(ibuf));
			free(rbuf);
			return NULL;
		}
		/*Allocating memory for double block case*/
		amtPerDevice = (iFrames + numDevs - 1) / numDevs;
		long long framecount = 0;
		doubleBlock = true;
		for(int i = 0; i < numDevs; i++){
			hipSetDevice(i);
			//theoretically should be 4. dividing by 8 to be conservative
			size_t freeSize = getFreeSize() / 4;
			freeSize -= rFrames;
			freeSize = pow(2, floor(log2((double)freeSize)));
			int currFrames = amtPerDevice;
			if (currFrames + M > freeSize){
				fprintf(stderr, "WARNING: One GPU has very little memory left. Redistributing memory.\n");
				currFrames = freeSize - M;
				amtPerDevice = iFrames;
			}
			
			if(framecount + currFrames > iFrames){
				currFrames = iFrames - framecount;
			}
			
			if(currFrames == 0){
				inSizes[i] = 0;
				continue;
			}
			inSizes[i] = currFrames + M;
			checkCudaErrors(hipMalloc(&d_ibufs[i], inSizes[i] * sizeof(float)));
			checkCudaErrors(hipMalloc(&d_rbufs[i], rFrames * sizeof(float)));
			framecount += currFrames;
			if(framecount >= iFrames) break;
		}
	}
	
	hipStream_t stream[numDevs];
	/**
	{
	TODO: Peer-to-Peer memcpy of rbuf
	hipDeviceProp_t prop;
	for(int i = 0; i )
	checkCudaErrors(hipGetDeviceProperties(&prop, ))
	int rbufDevNum = 0;
	for(int i = 0; i < numDevs; i++){
		for(int j = 0; j < numDevs; j++){
			if (i == j) continue;
			int num = 0;
			hipDeviceCanAccessPeer(&num, i, j);
			if(num){
				hipMemcpyAsync(d_rbufs[i], rbuf, rFrames * sizeof(float));
				rbufDevNum = i;
			}
		}
	}
	**/
	long long blockSize = 512;
	int numBlocks;
	/*Copy each chunk of input into each GPU and pad with 0's*/
	frames = 0;
	//fprintf(stderr, "%s Block\n", doubleBlock ? "Double" : "Single");
	for(int i = 0; i < numDevs; i++){
		hipSetDevice(i);
		checkCudaErrors(hipStreamCreate(&stream[i]));
		long long amtRead = inSizes[i] - M;
		if (frames + amtRead > iFrames){
			amtRead = iFrames - frames;
		}
		checkCudaErrors(hipMemcpyAsync(d_ibufs[i], ibuf + frames, amtRead * sizeof(float), hipMemcpyHostToDevice, stream[i]));
	
		numBlocks = (inSizes[i] - amtRead - 1 + blockSize) / blockSize;
		FillWithZeros<<<numBlocks, blockSize>>>(d_ibufs[i], amtRead, inSizes[i]);
		//fillWithZeroes(&d_ibufs[i], amtRead, inSizes[i]);
		if(!doubleBlock){	
			//fprintf(stderr, "Filling rbuf with zeroes to pad\n");
			numBlocks = (inSizes[i] - rFrames - 1 + blockSize) / blockSize;
			FillWithZeros<<<numBlocks, blockSize>>>(d_rbufs[i], rFrames, inSizes[i]);
			//fillWithZeroes(&d_rbufs[i], rFrames, inSizes[i]);
		}
		/*WILL BE REPLACED LATER*/
		checkCudaErrors(hipMemcpyAsync(d_rbufs[i], rbuf, rFrames * sizeof(float), hipMemcpyHostToDevice, stream[i]));
		//fprintf(stderr, "Copying reverb\n");
		//checkCudaErrors(hipMemcpy(d_rbufs[i], rbuf, rFrames * sizeof(float), hipMemcpyHostToDevice));
		///////////////////////////////////////////////////////////
		frames += inSizes[i] - M;
		
		if (frames >= iFrames){
			break;
		}
	}
	checkCudaErrors(hipHostFree(ibuf));
	free(rbuf);
	checkCudaErrors(hipSetDevice(0));
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	/*Loop through all input buffers and find the peak*/
	frames = iFrames;
	float minmax1 = 0;
	for(int i = 0 ; i < numDevs; i++){
		hipSetDevice(i);
		if(frames < 0) break;
		frames -= inSizes[i] - M;
		float minmax = DExtrema(d_ibufs[i], inSizes[i]);
		if(minmax > minmax1)
			minmax1 = minmax;
	}
	/*Convolve all chunks*/
	frames = iFrames;

	if(doubleBlock){
		for(int i = 0; i < numDevs; i++){
			hipSetDevice(i);
			blockConvolve(&d_ibufs[i], &d_rbufs[i], inSizes[i] - M, rFrames);
		}
	}
	else{
		//fprintf(stderr, "Single Block Convolution\n");
		for(int i = 0; i < numDevs; i++){
			hipSetDevice(i);
			if(frames < 0) break;
			frames -= inSizes[i] - M;
			convolve(&d_ibufs[i], &d_rbufs[i], &d_Cbufs[i], inSizes[i]);
		}
	}


	/*Overlap-add method to combine the convolved chunks*/
	
	int singleDev = 0;
	// size_t maxFree = 0;
	// for(int i = 0; i < numDevs; i++){
	// 	hipSetDevice(i);
	// 	if (maxFree < getFreeSize()){
	// 		maxFree = getFreeSize();
	// 		singleDev = i;
	// 	}
	// }
	hipSetDevice(singleDev);
	float *d_scratchSpace;
	checkCudaErrors(hipHostMalloc(&obuf, oFrames * sizeof(float)));
	checkCudaErrors(hipMalloc(&d_obuf, oFrames * sizeof(float)));
	checkCudaErrors(hipMemcpy(d_obuf, d_ibufs[0], inSizes[0] * sizeof(float), hipMemcpyDefault));
	checkCudaErrors(hipMalloc(&d_scratchSpace, M * sizeof(float)));
	hipSetDevice(0);
	checkCudaErrors(hipFree(d_ibufs[0]));
	
	long long size = inSizes[0];
	for(int i = 1; i < numDevs; i++){
		long long cpyAmount = inSizes[i] - M;
		if (size + cpyAmount > iFrames) {
			cpyAmount = oFrames - size;
		}
		hipSetDevice(i);
		checkCudaErrors(hipMemcpyAsync(d_obuf + size, d_ibufs[i] + M , cpyAmount * sizeof(float), hipMemcpyDefault, stream[i]));
		checkCudaErrors(hipMemcpy(d_scratchSpace, d_ibufs[i], M * sizeof(float), hipMemcpyDefault));
		
		hipSetDevice(singleDev);
		numBlocks = (M + blockSize - 1) / blockSize;
		PointwiseAdd <<< numBlocks, blockSize, 0, stream[0] >>>(d_scratchSpace, d_obuf + size - M, M);
		
		size += inSizes[i] - M;
		if(size >= oFrames){
			break;
		}	
	}
	frames = iFrames;
	for(int i = 0; i < numDevs; i++){
		if(frames < 0) break;
		frames -= inSizes[i] - M;
		checkCudaErrors(hipSetDevice(i));
		checkCudaErrors(hipStreamSynchronize(stream[i]));
		checkCudaErrors(hipStreamDestroy(stream[i]));
		if(i != 0)checkCudaErrors(hipFree(d_ibufs[i]));
	}
	
	hipSetDevice(singleDev);
	float minmax2;
	minmax2 = DExtrema(d_obuf, oFrames);
	float scale = minmax1/minmax2;
	
	int strides = 1;
	blockSize = 128;
	numBlocks = (oFrames / strides + blockSize - 1) / blockSize;
	while (numBlocks >(2U << 31 - 1)) {
		numBlocks = (oFrames / ++strides + blockSize - 1) / blockSize;
	}
	
	
	int nStreams = 4;
	int streamSize = (oFrames + nStreams - 1) / nStreams;
	int streamBytes = streamSize * sizeof(float);
	hipStream_t streams1[nStreams];
	for (int i = 0; i < nStreams; ++i) {
		checkCudaErrors(hipStreamCreate(&streams1[i]));
	}

	
	/*Scale + copy 4x*/
	numBlocks = (streamSize + blockSize - 1) / blockSize;
	for (int i = 0; i < nStreams; ++i) {
		int offset = i * streamSize;
		RealFloatScaleConcurrent << < numBlocks, blockSize, 0, streams1[i] >> > (d_obuf, oFrames, streamSize, scale, offset);
		if ( i == nStreams - 1){
			streamBytes = (oFrames - offset) * sizeof(float);
		}
		checkCudaErrors(hipMemcpyAsync(&obuf[offset], &d_obuf[offset], streamBytes, hipMemcpyDeviceToHost, streams1[i]));
	}
	for(int i = 0; i < 4; i++){
		checkCudaErrors(hipStreamDestroy(streams1[i]));
	}
	checkCudaErrors(hipSetDevice(0));
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	fprintf(stderr,"Time for GPU convolution: %f ms\n", milliseconds);
	checkCudaErrors(hipFree(d_obuf));
	return obuf;

}
